#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#define CUDA_SAFE_CALL_NO_SYNC( call) do {                              \
  hipError_t err = call;                                                 \
  if( hipSuccess != err) {                                             \
    fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",       \
                __FILE__, __LINE__, hipGetErrorString( err) );         \
    exit(EXIT_FAILURE);                                                 \
    } } while (0)

#define CUDA_SAFE_CALL( call) do {                                      \
  CUDA_SAFE_CALL_NO_SYNC(call);                                         \
  hipError_t err = hipDeviceSynchronize();                              \
  if( hipSuccess != err) {                                             \
     fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",      \
                 __FILE__, __LINE__, hipGetErrorString( err) );        \
     exit(EXIT_FAILURE);                                                \
     } } while (0)


//input element size
const int N = 1024*512;
//block size
const int blocksize = 1024;


__global__ void maxBandwidth(int n, float* in, float* out){
  
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if(i < n){
    float temp = in[i] + i * 2.0f;
    out[i] = out[i] + temp / (0.5f);
    
  }
}



int main(int argc, char **argv)
{
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  

  //unsigned int num_threads = N;
  unsigned int numbytes = N * sizeof(float);
  
  //allocate host memory
  float *in = (float *) malloc(numbytes);
  float *out =(float *) malloc(numbytes);

  
  // initalize the memory
  for( unsigned int i = 0; i < N ; ++i)
    {
        in[i] = (float)i;
	out[i] = 3.0f;
    }

  //allocate device memory
  float *d_in, *d_out;
  CUDA_SAFE_CALL(hipMalloc(&d_in, numbytes));
  CUDA_SAFE_CALL(hipMalloc(&d_out, numbytes));
  CUDA_SAFE_CALL(hipMemcpy(d_in, in, numbytes, hipMemcpyHostToDevice));



  dim3  block(N/blocksize, 1, 1);
    //max block size(1024, 1024, 64)
  dim3  thread(blocksize, 1 ,1);

  // execute the kernel
  hipEventRecord(start, 0);
  maxBandwidth<<< block, thread, numbytes>>>(N, d_in, d_out);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  // check if kernel execution generated and error
  // CUT_CHECK_ERROR("Kernel execution failed");


  CUDA_SAFE_CALL( hipMemcpy(out, d_out, numbytes, hipMemcpyDeviceToHost));


  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf("\nProcessing time: %f (ms)\n", elapsedTime);
  printf("Effective Bandwidth (GB/s): %f\n", (numbytes*3)/elapsedTime/1e6);
  // printf("Total number of memory read/write on GPU (bytes): %d\n\n", numbytes);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // cleanup memory
  free(in);
  free(out);
  CUDA_SAFE_CALL(hipFree(d_in));
  CUDA_SAFE_CALL(hipFree(d_out));

}
